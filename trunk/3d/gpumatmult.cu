
#include <hip/hip_runtime.h>
const int threadsPerBlock = 128;

unsigned int hostOffsetSize;
unsigned int hostTargetSize;
unsigned int hostSourceSize;

static unsigned int is_set=0;
static unsigned int deviceOffsetSize;
static unsigned int deviceTargetSize;
static unsigned int deviceSourceSize;

static int   *deviceOffset;
static float *deviceTargetX;
static float *deviceTargetY;
static float *deviceTargetZ;
static float *deviceTargetW;
static float *deviceSourceX;
static float *deviceSourceY;
static float *deviceSourceZ;
static float *deviceSourceG;

__global__ void kernel(int* deviceOffset, float* deviceTargetX, float* deviceTargetY, float* deviceTargetZ, float* deviceTargetW,
                       float sigma,       float* deviceSourceX, float* deviceSourceY, float* deviceSourceZ, float* deviceSourceG)
{
  int i = blockIdx.x * threadsPerBlock + threadIdx.x;
  int jbase,jsize,jblok,j,jb,jj;
  float targetX,targetY,targetZ,targetW,dx,dy,dz,coef;
  __shared__ float sharedSourceX[threadsPerBlock];
  __shared__ float sharedSourceY[threadsPerBlock];
  __shared__ float sharedSourceZ[threadsPerBlock];
  __shared__ float sharedSourceG[threadsPerBlock];

  targetX = deviceTargetX[i];
  targetY = deviceTargetY[i];
  targetZ = deviceTargetZ[i];
  targetW = 0;
  coef = 0.5f/(sigma*sigma);
  jbase = deviceOffset[blockIdx.x];
  jsize = deviceOffset[blockIdx.x+1]-deviceOffset[blockIdx.x];
  jblok = (jsize + threadsPerBlock - 1) / threadsPerBlock;
  for (j = 0; j < jblok-1; j++) {
    jb = jbase + j * threadsPerBlock + threadIdx.x;
    __syncthreads();
    sharedSourceX[threadIdx.x] = deviceSourceX[jb];
    sharedSourceY[threadIdx.x] = deviceSourceY[jb];
    sharedSourceZ[threadIdx.x] = deviceSourceZ[jb];
    sharedSourceG[threadIdx.x] = deviceSourceG[jb];
    __syncthreads();
#pragma unroll 32
    for(jj = 0; jj < threadsPerBlock; jj++){
      dx = targetX-sharedSourceX[jj];
      dy = targetY-sharedSourceY[jj];
      dz = targetZ-sharedSourceZ[jj];
      targetW += sharedSourceG[jj]*exp(-(dx*dx+dy*dy+dz*dz)*coef);
    }
  }
  jb = jbase + j * threadsPerBlock + threadIdx.x;
  __syncthreads();
  sharedSourceX[threadIdx.x] = deviceSourceX[jb];
  sharedSourceY[threadIdx.x] = deviceSourceY[jb];
  sharedSourceZ[threadIdx.x] = deviceSourceZ[jb];
  sharedSourceG[threadIdx.x] = deviceSourceG[jb];
  __syncthreads();
  for(jj = 0; jj < jsize - (j * threadsPerBlock); jj++){
    dx = targetX-sharedSourceX[jj];
    dy = targetY-sharedSourceY[jj];
    dz = targetZ-sharedSourceZ[jj];
    targetW += sharedSourceG[jj]*exp(-(dx*dx+dy*dy+dz*dz)*coef);
  }
  deviceTargetW[i] = targetW/M_PI*coef;
}

void gpumatmult(float *hostTargetX, float *hostTargetY, float *hostTargetZ, float *hostTargetW,
                float *hostSourceX, float *hostSourceY, float *hostSourceZ, float *hostSourceG,
                int *hostOffset, int iblok, float sigma, int numCluster, int numTrunc)
{
  hostOffsetSize = sizeof(int) * (numCluster+1);
  hostTargetSize = sizeof(float) * numCluster * threadsPerBlock;
  hostSourceSize = sizeof(float) * numCluster * numTrunc;

  if (is_set==0) {
    hipSetDevice(0);
    is_set=1;
  }
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) hipFree(deviceOffset);
    hipMalloc((void**)&deviceOffset,hostOffsetSize);
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostTargetSize>deviceTargetSize) {
    if(deviceTargetSize!=0) {
      hipFree(deviceTargetX);
      hipFree(deviceTargetY);
      hipFree(deviceTargetZ);
      hipFree(deviceTargetW);
    }
    hipMalloc((void**)&deviceTargetX,hostTargetSize);
    hipMalloc((void**)&deviceTargetY,hostTargetSize);
    hipMalloc((void**)&deviceTargetZ,hostTargetSize);
    hipMalloc((void**)&deviceTargetW,hostTargetSize);
    deviceTargetSize=hostTargetSize;
  }
  if (hostSourceSize>deviceSourceSize) {
    if(deviceSourceSize!=0) {
      hipFree(deviceSourceX);
      hipFree(deviceSourceY);
      hipFree(deviceSourceZ);
      hipFree(deviceSourceG);
    }
    hipMalloc((void**)&deviceSourceX,hostSourceSize);
    hipMalloc((void**)&deviceSourceY,hostSourceSize);
    hipMalloc((void**)&deviceSourceZ,hostSourceSize);
    hipMalloc((void**)&deviceSourceG,hostSourceSize);
    deviceSourceSize=hostSourceSize;
  }

  hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceTargetX,hostTargetX,hostTargetSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceTargetY,hostTargetY,hostTargetSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceTargetZ,hostTargetZ,hostTargetSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceSourceX,hostSourceX,hostSourceSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceSourceY,hostSourceY,hostSourceSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceSourceZ,hostSourceZ,hostSourceSize,hipMemcpyHostToDevice);
  hipMemcpy(deviceSourceG,hostSourceG,hostSourceSize,hipMemcpyHostToDevice);

  dim3 block(threadsPerBlock);
  dim3 grid(iblok);
  kernel<<< grid, block >>>(deviceOffset,deviceTargetX,deviceTargetY,deviceTargetZ,deviceTargetW,
                                   sigma,deviceSourceX,deviceSourceY,deviceSourceZ,deviceSourceG);

  hipMemcpy(hostTargetW,deviceTargetW,hostTargetSize,hipMemcpyDeviceToHost);

}
