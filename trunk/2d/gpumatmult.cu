
#include <hip/hip_runtime.h>
extern const int threadsPerBlock;

void gpumatmult(double *targetX, double *targetY, double *targetW,
                double *sourceX, double *sourceY, double *sourceG,
                int *offset, int iblok, double sigma)
{
  int ic,i,im,j;
  double dx,dy,w;

  for (ic=0; ic<iblok; ic++) {
    for (i=0; i<threadsPerBlock; i++) {
      im = ic*threadsPerBlock+i;
      w = 0;
      for (j=offset[ic]; j<offset[ic+1]; j++) {
        dx = targetX[im]-sourceX[j];
        dy = targetY[im]-sourceY[j];
        w += sourceG[j]*exp(-(dx*dx+dy*dy)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
      }
      targetW[im] = w;
    }
  }
}
