#include "hip/hip_runtime.h"
#include <cutil.h>
const int threadsPerBlock = 128;

unsigned int hostOffsetSize;
unsigned int hostTargetSize;
unsigned int hostSourceSize;

static unsigned int is_set=0;
static unsigned int deviceOffsetSize;
static unsigned int deviceTargetSize;
static unsigned int deviceSourceSize;

static int   *deviceOffset;
static float *deviceTargetX;
static float *deviceTargetY;
static float *deviceTargetW;
static float *deviceSourceX;
static float *deviceSourceY;
static float *deviceSourceG;

__global__ void kernel(int* deviceOffset, float* deviceTargetX, float* deviceTargetY, float* deviceTargetW,
                       float sigma,       float* deviceSourceX, float* deviceSourceY, float* deviceSourceG)
{
  int i = blockIdx.x * threadsPerBlock + threadIdx.x;
  int jbase,jsize,jblok,j,jb,jj;
  float targetX,targetY,targetW,dx,dy,coef;
  __shared__ float sharedSourceX[threadsPerBlock];
  __shared__ float sharedSourceY[threadsPerBlock];
  __shared__ float sharedSourceG[threadsPerBlock];

  targetX = deviceTargetX[i];
  targetY = deviceTargetY[i];
  targetW = 0;
  coef = 0.5f/(sigma*sigma);
  jbase = deviceOffset[blockIdx.x];
  jsize = deviceOffset[blockIdx.x+1]-deviceOffset[blockIdx.x];
  jblok = (jsize + threadsPerBlock - 1) / threadsPerBlock;
  for (j = 0; j < jblok-1; j++) {
    jb = jbase + j * threadsPerBlock + threadIdx.x;
    __syncthreads();
    sharedSourceX[threadIdx.x] = deviceSourceX[jb];
    sharedSourceY[threadIdx.x] = deviceSourceY[jb];
    sharedSourceG[threadIdx.x] = deviceSourceG[jb];
    __syncthreads();
#pragma unroll 32
    for(jj = 0; jj < threadsPerBlock; jj++){
      dx = targetX-sharedSourceX[jj];
      dy = targetY-sharedSourceY[jj];
      targetW += sharedSourceG[jj]*exp(-(dx*dx+dy*dy)*coef);
    }
  }
  jb = jbase + j * threadsPerBlock + threadIdx.x;
  __syncthreads();
  sharedSourceX[threadIdx.x] = deviceSourceX[jb];
  sharedSourceY[threadIdx.x] = deviceSourceY[jb];
  sharedSourceG[threadIdx.x] = deviceSourceG[jb];
  __syncthreads();
  for(jj = 0; jj < jsize - (j * threadsPerBlock); jj++){
    dx = targetX-sharedSourceX[jj];
    dy = targetY-sharedSourceY[jj];
    targetW += sharedSourceG[jj]*exp(-(dx*dx+dy*dy)*coef);
  }
  deviceTargetW[i] = targetW/M_PI*coef;
}

void gpumatmult(float *hostTargetX, float *hostTargetY, float *hostTargetW,
                float *hostSourceX, float *hostSourceY, float *hostSourceG,
                int *hostOffset, int iblok, float sigma, int numCluster, int numTrunc)
{
  hostOffsetSize = sizeof(int) * (numCluster+1);
  hostTargetSize = sizeof(float) * numCluster * threadsPerBlock;
  hostSourceSize = sizeof(float) * numCluster * numTrunc;

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostTargetSize>deviceTargetSize) {
    if(deviceTargetSize!=0) {
      CUDA_SAFE_CALL(hipFree(deviceTargetX));
      CUDA_SAFE_CALL(hipFree(deviceTargetY));
      CUDA_SAFE_CALL(hipFree(deviceTargetW));
    }
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceTargetX,hostTargetSize));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceTargetY,hostTargetSize));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceTargetW,hostTargetSize));
    deviceTargetSize=hostTargetSize;
  }
  if (hostSourceSize>deviceSourceSize) {
    if(deviceSourceSize!=0) {
      CUDA_SAFE_CALL(hipFree(deviceSourceX));
      CUDA_SAFE_CALL(hipFree(deviceSourceY));
      CUDA_SAFE_CALL(hipFree(deviceSourceG));
    }
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceSourceX,hostSourceSize));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceSourceY,hostSourceSize));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceSourceG,hostSourceSize));
    deviceSourceSize=hostSourceSize;
  }

  CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(deviceTargetX,hostTargetX,hostTargetSize,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(deviceTargetY,hostTargetY,hostTargetSize,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(deviceSourceX,hostSourceX,hostSourceSize,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(deviceSourceY,hostSourceY,hostSourceSize,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(deviceSourceG,hostSourceG,hostSourceSize,hipMemcpyHostToDevice));

  dim3 block(threadsPerBlock);
  dim3 grid(iblok);
  kernel<<< grid, block >>>(deviceOffset,deviceTargetX,deviceTargetY,deviceTargetW,
                                   sigma,deviceSourceX,deviceSourceY,deviceSourceG);
  CUT_CHECK_ERROR("Kernel execution failed");

  CUDA_SAFE_CALL(hipMemcpy(hostTargetW,deviceTargetW,hostTargetSize,hipMemcpyDeviceToHost));

}
